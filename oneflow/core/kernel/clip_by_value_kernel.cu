#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void GpuForward(const int64_t elem_cnt, const T* in_ptr, const T min_val,
                           const T max_val, T* out_ptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) { out_ptr[i] = min(max(in_ptr[i], min_val), max_val); }
}

template<typename T>
__global__ void GpuBackward(const int64_t elem_cnt, const T* dy_ptr, const T* x_ptr,
                            const T min_val, const T max_val, T* dx_ptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    dx_ptr[i] = dy_ptr[i] * static_cast<T>(x_ptr[i] > min_val && x_ptr[i] < max_val);
  }
}

}  // namespace

template<typename T>
class ClipByValueGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ClipByValueGpuKernel);
  ClipByValueGpuKernel() = default;
  ~ClipByValueGpuKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* in_blob = BnInOp2Blob("in");
    Blob* out_blob = BnInOp2Blob("out");

    const auto& shape = in_blob->shape();
    CHECK(out_blob->shape() == shape);
    const ClipByValueOpConf& conf = this->op_conf().clip_by_value_conf();
    const T min_val = conf.has_min_val() ? static_cast<T>(conf.min_val()) : GetMinVal<T>();
    const T max_val = conf.has_max_val() ? static_cast<T>(conf.max_val()) : GetMaxVal<T>();
    GpuForward<<<BlocksNum4ThreadsNum(shape.elem_cnt()), kCudaThreadsNumPerBlock, 0,
                 ctx.device_ctx->hip_stream()>>>(shape.elem_cnt(), in_blob->dptr<T>(), min_val,
                                                  max_val, out_blob->mut_dptr<T>());
  }
};

template<typename T>
class ClipByValueGradGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(ClipByValueGradGpuKernel);
  ClipByValueGradGpuKernel() = default;
  ~ClipByValueGradGpuKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* dy_blob = BnInOp2Blob("dy");
    const Blob* x_blob = BnInOp2Blob("x");
    Blob* dx_blob = BnInOp2Blob("dx");

    const auto& shape = dy_blob->shape();
    CHECK(dx_blob->shape() == shape);
    const ClipByValueGradOpConf& conf = this->op_conf().clip_by_value_grad_conf();
    const T min_val = conf.has_min_val() ? static_cast<T>(conf.min_val()) : GetMinVal<T>();
    const T max_val = conf.has_max_val() ? static_cast<T>(conf.max_val()) : GetMaxVal<T>();
    GpuBackward<<<BlocksNum4ThreadsNum(shape.elem_cnt()), kCudaThreadsNumPerBlock, 0,
                  ctx.device_ctx->hip_stream()>>>(shape.elem_cnt(), dy_blob->dptr<T>(),
                                                   x_blob->dptr<T>(), min_val, max_val,
                                                   dx_blob->mut_dptr<T>());
  }
};

#define MAKE_ENTRY(type_cpp, type_proto)                                                      \
  NEW_REGISTER_KERNEL(OperatorConf::kClipByValueConf, ClipByValueGpuKernel<type_cpp>)         \
      .SetIsMatchedPred([](const KernelConf& conf) {                                          \
        return (DeviceType::kGPU == conf.op_attribute().op_conf().device_type())              \
               && (type_proto == conf.data_type());                                           \
      });                                                                                     \
  NEW_REGISTER_KERNEL(OperatorConf::kClipByValueGradConf, ClipByValueGradGpuKernel<type_cpp>) \
      .SetIsMatchedPred([](const KernelConf& conf) {                                          \
        return (DeviceType::kGPU == conf.op_attribute().op_conf().device_type())              \
               && (type_proto == conf.data_type());                                           \
      });

OF_PP_FOR_EACH_TUPLE(MAKE_ENTRY, ARITHMETIC_DATA_TYPE_SEQ);

}  // namespace oneflow
